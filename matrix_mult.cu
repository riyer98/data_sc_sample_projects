#include<iostream>
#include<__clang_cuda_runtime_wrapper.h>
#include<hip/hip_runtime.h>


__global__ void kernel() {
    printf("Hello from the GPU!\n");
}

int main() {
    kernel<<<1, 1>>>();
    hipDeviceSynchronize();
    return 0;
}

